#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>   // for rand(), srand()
#include <time.h>     // for time(NULL)
#include <math.h>     // for sqrtf
#include "neural_network.h"
#include "utils.h"

// Data Structures
typedef struct {
    int inputSize;
    int outputSize;
    float *weights;
    float *biases;
    float *d_weights;
    float *d_biases;
} Layer;

typedef struct {
    int numLayers;
    Layer *layers;
    float learningRate;
} NeuralNetwork;

// Initializations
void initLayer(Layer *layer, int inputSize, int outputSize) {
    
    // Store dimensions
    layer->inputSize = inputSize;
    layer->outputSize = outputSize;

    // Calculate parameter sizes
    size_t weightBytes = inputSize * outputSize * sizeof(float);
    size_t biasBytes = outputSize * sizeof(float);

    // Allocate GPU memory
    checkCudaError(hipMalloc((void **)&(layer->weights), weightBytes), "hipMalloc layer->weights");
    checkCudaError(hipMalloc((void **)&(layer->biases), biasBytes), "hipMalloc layer->biases");
    checkCudaError(hipMalloc((void **)&(layer->d_weights), weightBytes), "hipMalloc layer->d_weights");
    checkCudaError(hipMalloc((void **)&(layer->d_biases), biasBytes), "hipMalloc layer->d_biases");

    // Allocate temporary host arrays for initialization
    float *h_weights = (float *)malloc(weightBytes);
    float *h_biases = (float *)malloc(biasBytes);

    if (!h_weights || !h_biases)
    {
        fprintf(stderr, "Host memory allocation failed in initLayer\n");
        exit(EXIT_FAILURE);
    }

    // Seed the random generator
    static int seedInitialized = 0;
    if (!seedInitialized)
    {
        srand((unsigned int)time(NULL));
        seedInitialized = 1;
    }

    // Simple random init -- Maybe make more sophisticated?
    float stddev = sqrtf(2.0f / (float)inputSize);
    for (int i = 0; i < inputSize * outputSize; i++)
    {
        // random float in [-1,1]
        float r = ((float)rand() / RAND_MAX) * 2.0f - 1.0f;
        h_weights[i] = r * stddev;
    }

    // Initialize biases to zero (or small random)
    for (int i = 0; i < outputSize; i++)
    {
        h_biases[i] = 0.0f;
    }

    // Copy from host to device
    checkCudaError(hipMemcpy(layer->weights, h_weights, weightBytes, hipMemcpyHostToDevice), "hipMemcpy weights");
    checkCudaError(hipMemcpy(layer->biases, h_biases, biasBytes, hipMemcpyHostToDevice), "hipMemcpy biases");

    // Initialize gradients to zero
    checkCudaError(hipMemset(layer->d_weights, 0, weightBytes), "hipMemset d_weights");
    checkCudaError(hipMemset(layer->d_biases, 0, biasBytes), "hipMemset d_biases");

    // Free host memory
    free(h_weights);
    free(h_biases);
}

void initNetwork(NeuralNetwork *network, int *layerSizes, int numLayers, float learningRate) {
    // Typically, numLayers might be the count of the *layerSizes array* - 1 if it includes input.
    // E.g.: layerSizes = [784, 128, 10] => numLayers = 2 fully-connected layers.
    // We'll interpret numLayers as "number of layers" in the sense of how many times we call initLayer.

    network->numLayers = numLayers - 1; // e.g. if layerSizes has 3 elements => 2 layers
    network->learningRate = learningRate;

    // Allocate host array for the layers
    network->layers = (Layer *)malloc(network->numLayers * sizeof(Layer));
    if (!network->layers)
    {
        fprintf(stderr, "Host memory allocation failed in initNetwork\n");
        exit(EXIT_FAILURE);
    }

    // Initialize each layer
    for (int i = 0; i < network->numLayers; i++) {
        int inSize = layerSizes[i];
        int outSize = layerSizes[i + 1];

        initLayer(&(network->layers[i]), inSize, outSize);
    }

    // Print Summary Info
    printf("[initNetwork] Created %d layers.\n", network->numLayers);
    for (int i = 0; i < network->numLayers; i++)
    {
        printf("  Layer %d: inputSize=%d, outputSize=%d\n",
               i, network->layers[i].inputSize, network->layers[i].outputSize);
    }
}

void freeLayer(Layer *layer) {
    if (layer->weights)
        hipFree(layer->weights);
    if (layer->biases)
        hipFree(layer->biases);
    if (layer->d_weights)
        hipFree(layer->d_weights);
    if (layer->d_biases)
        hipFree(layer->d_biases);

    layer->weights = NULL;
    layer->biases = NULL;
    layer->d_weights = NULL;
    layer->d_biases = NULL;
}

void freeNetwork(NeuralNetwork *network)
{
    for (int i = 0; i < network->numLayers; i++)
    {
        freeLayer(&(network->layers[i]));
    }
    free(network->layers);
    network->layers = NULL;
}
